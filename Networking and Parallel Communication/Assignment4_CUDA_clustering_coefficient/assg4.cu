#include "hip/hip_runtime.h"
/*
Assignment 4 
Thomas Kistler
11/7/17
*/

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <vector>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using namespace std;

__shared__ float SUM;//overall sum

int MAX;//max node

/**
 * CUDA Kernel Device code
 *
 * Computes the clustering coefficient of vector adjVector
 */
__global__ void clusterCoefficient(int * adjVector, int numElements) {
   
   int index = threadIdx.x + blockIdx.x * blockDim.x;
   float result = 0;
   float Ni = 0;
   float Mi = 0;
   vector<int> neighbors;
   //calculate degree (Ni)
   for(int i = index; i < index + MAX; i++)
   {
	if(adjVector[index] == 1)
	{
	   Ni += 1.0;//add to degree of current node	
	   //get neighbors list during degree count
           neighbors.push_back(index);	
	}
   }
   //calculate edges between neighbors (Mi)
   for(int i = 0; i < neighbors.size(); i++)
   {
	for(int j = 0; j < neighbors.size(); j++)
	{
	    /* add 1 to the edge count if the two neighbors share an edge 
	       meaning there's a 1 at the correct index                   */
	    if( i != j && adjVector[ neighbors[j] * MAX + neighbors[i] ] == 1)
		    Mi += 1.0;
	}
   }

   //calculate clustering coefficient
   float result = (2 * Mi) / (Ni * (Ni - 1)); 
   atomicAdd(SUM, result);//add result, atomically
}

/**
 * Host main routine
 */
int main(int argc, char* argv[]){
    
    /* Read in file */
    cout<<"Please enter the input file name"<<endl;
    string infile;
    cin>>infile;
    fstream myfile(infile, std::ios_base::in);
    int u, v;
    int maxNode = 0;
    vector<pair<int,int> > allEdges;
    while(myfile >> u >> v)
    {
        allEdges.push_back(make_pair(u,v));
        if(u > maxNode)
          maxNode = u;

        if(v > maxNode)
          maxNode = v;                 
    }

    MAX = maxNode;
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int threadsPerBlock = 1;
    int numElements = maxNode * maxNode;
    cout<<"Graph has " << numElements <<" nodes "<<endl;

    size_t size = numElements * sizeof(int);

    // Allocate the host input vector A
    int *h_A = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL )
    {
        fprintf(stderr, "Failed to allocate host vector!\n");
        exit(EXIT_FAILURE);
    }
    
    //populate the adjacency vector 
    for(int i = 0; i < allEdges.size(); i++){
       u = allEdges[i].first;
       v = allEdges[i].second;
       h_A[(i * u) + v] = 1;  //using 1D vector instead of 2D matrix
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vector into host memory to the device 
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int blocksPerGrid = numElements;//assuming threadsPerBlock = 1
    //launch kernel
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    //call kernel
    clusterCoefficient<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch clusterCoefficient kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    //result
    float CC = SUM / numElements;
    cout<<endl<<"The clustering coefficient for the graph is: "<<cc<<endl<<endl;

    printf("Done\n");
    return 0;
}











